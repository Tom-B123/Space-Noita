#include "hip/hip_runtime.h"
// Holds RGB colour data for a pixel
struct Colour {
    // Stores the colour in a more readable format, chars used as channels must be between 0 and 31
    short r;
    short g;
    short b;
    bool a;
};

// Stores all the relevant data about a pixel and its behaviour
struct Tag {
    int     id;
    int     weight;
    bool    is_powder;
    bool    is_liquid;
    bool    is_gas;
    bool    is_flammable;
};

// Range of colours corresponds to the ID
struct Bound {
    int min_r;
    int max_r;
    int min_g;
    int max_g;
    int min_b;
    int max_b;
    int id;
};

struct Pos {
    int x;
    int y;
};

int pos_to_index(struct Pos pos, int width) {
    return (pos.x+pos.y*width);
}

struct Colour get_pixel_colour(short pixel_data) {
    int i_pixel_data = (int)pixel_data;
    struct Colour out = {(short)(i_pixel_data & 63488) >> 11, (short)(i_pixel_data & 1984) >> 6, (short)(i_pixel_data & 62) >> 1, (bool)(pixel_data & 1) };

    if (out.r < 0) { out.r += 32; }

    return out;
}
short get_pixel_data(struct Colour pixel_colour) {
    short out = (short)(pixel_colour.a + (pixel_colour.b << 1) + (pixel_colour.g << 6) + (pixel_colour.r << 11));
    return out;
}

int get_material_id(struct Colour pixel_colour, struct Bound *bounds) {
    for (int i = 0; i < 2; i++) {
        //if (pixel_colour.r > 0) printf("min: %i, max: %i, pixel: %i \n",bounds[i].min_r,bounds[i].max_r,pixel_colour.r);
        if (
            pixel_colour.r >= bounds[i].min_r && pixel_colour.r <= bounds[i].max_r &&
            pixel_colour.g >= bounds[i].min_g && pixel_colour.g <= bounds[i].max_g &&
            pixel_colour.b >= bounds[i].min_b && pixel_colour.b <= bounds[i].max_b
        ) {
            return i;
        }
    }
    return -1;
}

struct Tag get_tags(struct Colour pixel_colour, struct Bound *bounds) {
    int material_id = get_material_id(pixel_colour,bounds);

    struct Tag out_tag = {material_id,0,false,false,false,false};
    switch(material_id) {
        case 0:
            // Sand
            out_tag.is_powder = true;
        default:
            break;
    }
    return out_tag;
}

bool is_falling(struct Pos pos, int width, struct Bound *bounds, __global const short *src_pos) {
    struct Tag tag = get_tags(get_pixel_colour(src_pos[pos_to_index(pos,width)]),bounds);
    return tag.is_powder || tag.is_liquid;
}

bool is_empty(struct Pos pos, int width, __global const short *src_pos) {
    struct Colour cell_colour = get_pixel_colour(src_pos[pos_to_index(pos,width)]);
    if (pos.x < 0 || pos.x > width - 1 || pos.y < 0) { return false; }
    return !cell_colour.a;
}

int falling_cell_priority(struct Pos pos, int width, struct Bound *bounds, __global const short *src_pos) {

    // Cell falling from above
    pos.y += 1;
    if (is_falling(pos,width,bounds,src_pos))           { return 0;}

    // Cell falling from right
    pos.x -= 1;
    if (is_falling(pos,width,bounds,src_pos))           { return 1;}

    // Cell falling from left
    pos.x+=2;
    if (is_falling(pos,width,bounds,src_pos))           { return 2;}

    // Cell flowing from right
    pos.y -= 1;
    pos.x -= 1;
    if (is_falling(pos,width,bounds,src_pos))           { return 3;}

    // Cell flowing from left
    pos.x += 2;
    if (is_falling(pos,width,bounds,src_pos))           { return 4;}

    return 4;
}

struct Pos update_powder(struct Pos pos, int width, int height, struct Bound *bounds, __global const short *src_pos) {
    // Check beneath
    pos.y -= 1;
    if (is_empty(pos,width,src_pos)) {
        return pos;
    }

    // Check right
    pos.x += 1;
    if (is_empty(pos,width,src_pos) && falling_cell_priority(pos,width,bounds,src_pos) >= 1) {
        return pos;
    }

    // Check left
    pos.x -= 2;
    if (is_empty(pos,width,src_pos) && falling_cell_priority(pos,width,bounds,src_pos) >= 1) {
        return pos;
    }
    pos.x += 1;
    pos.y += 1;
    return pos;
}

__kernel void sampleKernel(__global const short *src_pos, __global short *dst_pos, __global const float *src_gravity_angle, __global const int *src_world_dims, __global int *src_step) {
    const int gid = get_global_id(0);
    const int width = src_world_dims[0];
    const int height = src_world_dims[1];
    const int x = (gid) % width;
    const int y = (gid) / width;
    const int step = src_step[0];

    const struct Colour blank = { 0,0,0,0 };

    struct Pos pos = {x,y};

    struct Colour cell_colour = get_pixel_colour(src_pos[gid]);

    // Materials should have a range, eg. 2 in rgb. These colours correspond to unique colours, e.g. metal can be white / red etc.
    struct Bound bounds[] = {
        {29,31,29,31,0,0,0}, // Sand (ID 0)
        {0,0,0,2,29,31,1},   // Water (ID 1)
    };

    struct Tag tag = get_tags(cell_colour,&bounds);

    if (tag.is_powder) { pos = update_powder(pos,width,height,&bounds,src_pos); }

    int n_id = pos_to_index(pos,width);

    dst_pos[gid] = get_pixel_data(blank);
    dst_pos[n_id] = get_pixel_data(cell_colour);
    return;
};
