#include "hip/hip_runtime.h"
// Holds RGB colour data for a pixel
struct Colour {
    short r;
    short g;
    short b;
};

// Holds a position and a velocity to update both in 1 function
struct PosVel {
    int x;
    int y;
    int vel_magnitude;
    int vel_angle;
};

int get_weight(int material) {
    const int material_count = 3;

    // If the cell is invalid or an invalid material ID, return -1
    if (material < 0 || material > material_count) { return -1;}

    const int weight[material_count] = {
        30, // sand
        80, // silver
        10  // water
    };

    return weight[material];
}

// Returns PI and PI/2 for calculations
float PI() { return 3.14159f; }
float half_PI() { return 1.570795f; }

// Returns the constant acceleration values
float get_y_accel() { return -1.2f; }
float get_x_accel() { return 0.0f; }

// Convert from cartesian to 1D index
int pos_to_index(int x, int y, int width) {
    return 3 * (x + y * width);
}
int pos_to_vel_index(int x, int y, int width) {
    return (x + y * width);
}

// Get the cell colour
struct Colour get_cell(int x, int y, int width, int height, __global const short *a) {
    if (y < 0 || y > height -1 || x < 0 || x > width -1) {
        struct Colour out = {32767, 32767, 32767 };
        return out;
    }
    int index = pos_to_index(x,y,width);

    struct Colour n_colour = { a[index], a[index + 1], a[index + 2] };
    return n_colour;
}

int material_type(int x, int y, int width, int height, __global const short *src_pos) {
    int combined_rgb;
    const int r_set = 127 * 256 * 256;
    const int g_set = 127 * 256;
    const int b_set = 127;

    if (x < 0 || x > width - 1 || y < 0 || y > height - 1) { return -2; }

    struct Colour colour = get_cell(x,y,width,height,src_pos);
    combined_rgb = (int)(colour.b / 256) + (int)(colour.g / 256) * 256 + (int)(colour.r / 256) * 256 * 256;
    // -1 = empty, -2 = invalid
    switch (combined_rgb) {
        // Order: Powders, liquids, gasses, solids
        case g_set + r_set:
            // Sand
            return 0;
        case g_set + r_set + b_set:
            // Silver
            return 1;
        case b_set:
            // Water
            return 2;
        default:
            return -1;
    }
}

// Cell contents
bool is_empty(int x, int y, int width, int height, __global const short *src_pos) {
    if (x < 0 || x > width - 1 || y < 0 || y > height - 1) { return false; }
    return material_type(x,y,width,height,src_pos) == -1;
}
bool is_sand(int x, int y, int width, int height, __global const short *src_pos) {
    return material_type(x,y,width,height,src_pos) == 0;
}
bool is_silver(int x, int y, int width, int height, __global const short *src_pos) {
    return material_type(x,y,width,height,src_pos) == 1;
}
bool is_water(int x, int y, int width, int height, __global const short *src_pos) {
    return material_type(x,y,width,height,src_pos) == 2;
}
bool is_steam(int x, int y, int width, int height, __global const short *src_pos) {
    return false;//material_type(x,y,width,height,src_pos) == 2;
}

// Cell types
bool is_powder(int x, int y, int width, int height, __global const short *src_pos) {
    int t = material_type(x,y,width,height,src_pos);
    return t >= 0 && t <= 1;
}
bool is_liquid(int x, int y, int width, int height, __global const short *src_pos) {
    int t = material_type(x,y,width,height,src_pos);
    return t >= 2 && t <= 2;
}
bool is_gas(int x, int y, int width, int height, __global const short *src_pos) {
    int t = material_type(x,y,width,height,src_pos);
    return t >= 2 && t <= 2;
}

// Cell properties
bool is_falling(int x, int y, int width, int height, __global const short *a) {
    return is_liquid(x,y,width,height,a) || is_powder(x,y,width,height,a);
}

// Take the target position and return the highest priority cell (0->4, sand only uses 0->2)
int falling_cell_priority(int x, int y, int width, int height, __global const short *a) {
    if (is_falling(x,y+1,width,height,a))                                        { return 0; }
    if (is_falling(x-1,y+1,width,height,a))                                      { return 1; }
    if (is_falling(x+1,y+1,width,height,a))                                      { return 2; }
    if (is_liquid(x-1,y,width,height,a))                                         { return 3; }
    if (is_liquid(x+1,y,width,height,a))                                         { return 4; }
    return 4;
}

int rising_cell_priority(int x, int y, int width, int height, __global const short *a) {
    if (is_gas(x,y-1,width,height,a))                                            { return 0; }
    if (is_gas(x-1,y-1,width,height,a))                                          { return 1; }
    if (is_gas(x+1,y-1,width,height,a))                                          { return 2; }
    if (is_gas(x-1,y,width,height,a))                                            { return 3; }
    if (is_gas(x+1,y,width,height,a))                                            { return 4; }
    return 4;
}

// Process: cells have a flowing priority:
// 1 0 2
// 3 X 4
// 1 0 2
// Iterate through each check in this order.
// For priority 2, check no priority 1 cell is present (no cell is above the empty space)
// For priority 3, check no priority 2 or 1 cell is present.
// If any higher priority exists, do not move and they will move in


// Updating "static" particles, i.e. liquids or powers that are in a pool / stack, not free falling
struct PosVel update_powder(struct PosVel pos_vel, int width, int height, __global const short *src_pos) {
    // P0, always goes through
    if (is_empty(pos_vel.x,pos_vel.y-1,width,height,src_pos)) {
        pos_vel.y -= 1;
        return pos_vel;
    }
    // P1
    if (is_empty(pos_vel.x+1,pos_vel.y-1,width,height,src_pos) && falling_cell_priority(pos_vel.x+1,pos_vel.y-1,width,height,src_pos) >= 1) {
        pos_vel.x += 1;
        pos_vel.y -= 1;
        return pos_vel;
    }
    // P2
    if (is_empty(pos_vel.x-1,pos_vel.y-1,width,height,src_pos) && falling_cell_priority(pos_vel.x-1,pos_vel.y-1,width,height,src_pos) >= 2) {
        pos_vel.x -= 1;
        pos_vel.y -= 1;
        return pos_vel;
    }
    return pos_vel;
}

struct PosVel update_liquid(struct PosVel pos_vel, int width, int height, int step, __global const short *src_pos) {


    // P0, always goes through
    if (is_empty(pos_vel.x,pos_vel.y-1,width,height,src_pos)) {
        pos_vel.y -= 1;
    }
    // P1
    else if (is_empty(pos_vel.x+1,pos_vel.y-1,width,height,src_pos) && falling_cell_priority(pos_vel.x+1,pos_vel.y-1,width,height,src_pos) >= 1) {
        pos_vel.x += 1;
        pos_vel.y -= 1;
    }
    // P2
    else if (is_empty(pos_vel.x-1,pos_vel.y-1,width,height,src_pos) && falling_cell_priority(pos_vel.x-1,pos_vel.y-1,width,height,src_pos) >= 2) {
        pos_vel.x -= 1;
        pos_vel.y -= 1;
    }
    else {
        if (step % 2 == 0) {
            //P3
            if (is_empty(pos_vel.x+1,pos_vel.y,width,height,src_pos) && falling_cell_priority(pos_vel.x+1,pos_vel.y,width,height,src_pos) >= 3) {
                pos_vel.x += 1;
            }
            // P4
            else if (is_empty(pos_vel.x-1,pos_vel.y,width,height,src_pos) && falling_cell_priority(pos_vel.x-1,pos_vel.y,width,height,src_pos) >= 4) {
                pos_vel.x -= 1;
            }
        }
        else {
            //P4
            if (is_empty(pos_vel.x-1,pos_vel.y,width,height,src_pos) && falling_cell_priority(pos_vel.x-1,pos_vel.y,width,height,src_pos) >= 4) {
                pos_vel.x -= 1;
            }
            // P3
            else if (is_empty(pos_vel.x+1,pos_vel.y,width,height,src_pos) && falling_cell_priority(pos_vel.x+1,pos_vel.y,width,height,src_pos) >= 3) {
                pos_vel.x += 1;
            }
        }
    }
    if (pos_vel.x < 0) { pos_vel.x = 0; }
    if (pos_vel.x > width - 1) { pos_vel.x = width - 1; }
    if (pos_vel.y < 0) { pos_vel.y = 0; }
    if (pos_vel.y > height - 1) { pos_vel.y = height - 1; }

    return pos_vel;
}

int update_gas(int x, int y, int width, int height, __global const short *a) {
    // P0, always goes through
    if (is_empty(x,y+1,width,height,a)) {
        return pos_to_index(x,y+1,width);
    }
    // P1
    if (is_empty(x+1,y+1,width,height,a) && rising_cell_priority(x+1,y+1,width,height,a) >= 1) {
        return pos_to_index(x+1,y+1,width);
    }
    // P2
    if (is_empty(x-1,y+1,width,height,a) && rising_cell_priority(x-1,y+1,width,height,a) >= 2) {
        return pos_to_index(x-1,y+1,width);
    }
    // P3
    if (is_empty(x+1,y,width,height,a) && rising_cell_priority(x+1,y,width,height,a) >= 3) {
        return pos_to_index(x+1,y,width);
    }
    // P4
    if (is_empty(x-1,y,width,height,a) && rising_cell_priority(x-1,y,width,height,a) >= 4) {
        return pos_to_index(x-1,y,width);
    }
    return pos_to_index(x,y,width);
}

// Swap cell the cell above / below, depending on material weight
// Pseudo bubble sort to move any column from an unsorted mixture to a sorted one, lightest at the top and heaviest at the bottom
struct PosVel update_weight(struct PosVel pos_vel, const int width,const int height, int step, __global const short *src_pos) {

    // Plan: iteratively look above and below until a "stable" cell is found, ie two cells that don't want to swap.
    // Compare weight difference between self and above / below cells, greater difference = swap direction

    // Worst case = stack of differently weighted cells from very bottom to very top
    int weights[5];

    // Get cell weight
    weights[2] = get_weight(material_type(pos_vel.x,pos_vel.y,width,height,src_pos));
    // Get above and below weight
    weights[3] = get_weight(material_type(pos_vel.x,pos_vel.y+1,width,height,src_pos));
    weights[1] = get_weight(material_type(pos_vel.x,pos_vel.y-1,width,height,src_pos));

    // If above and below are invalid or the cell are already in order, return out
    if (weights[3] == -1 && weights[1] == -1 || (weights[1] >= weights[2] && weights[3] <= weights[2])) { return pos_vel; }

    // Direction to above unless below is the greater distance
    int direction = 0;
    int other_direction = 0;

    // If the rise priority is positive (the cell should rise) and more significant than the sink priority, direction to rising
    if (weights[3] - weights[2] > 0 && weights[3] - weights[2] > weights[2] - weights[1]) { direction = 1;}

    // If the sink priority is positive (the cell should sink) and more significant than the rise priority, direction to sinking
    if (weights[2] - weights[1] > 0 && weights[2] - weights[1] > weights[3] - weights[2]) { direction = -1;}

    // Confine the direction to not point out of bounds
    if (pos_vel.y + direction < 0 || pos_vel.y + direction > height - 1) { direction = 0; }


    // Add in the weight for the other cell's movement
    weights[2+direction+direction] = get_weight(material_type(pos_vel.x,pos_vel.y+direction+direction,width,height,src_pos));

    // If the rise priority is positive (the cell should rise) and more significant than the sink priority, direction to rising
    if (weights[3 + direction] - weights[2 + direction] > 0 && weights[3 + direction] - weights[2 + direction] > weights[2 + direction] - weights[1 + direction]) { other_direction = 1;}

    // If the sink priority is positive (the cell should sink) and more significant than the rise priority, direction to sinking
    if (weights[2 + direction] - weights[1 + direction] > 0 && weights[2 + direction] - weights[1 + direction] > weights[3 + direction] - weights[2 + direction]) { other_direction = -1;}

    // Confine the direction to not point out of bounds
    if (pos_vel.y + other_direction < 0 || pos_vel.y + other_direction > height - 1) { other_direction = 0; }


    // Ensure the other has the opposite direction to you

    // If directions are opposite, move that way
    if (direction == 1 && other_direction == -1) {
        // Ensure the weight difference is correct before swapping
        pos_vel.y += 1;
    }
    if (direction == -1 && other_direction == 1) {
        // Ensure the weight difference is correct before swapping
        pos_vel.y -= 1;
    }
    return pos_vel;
}


// Updating "dynamic" particles, i.e. liquids or powers that are free falling

// Get the angle back from a given x,y velocity
float get_angle(float x, float y) {
    if (x >= 0 && y >= 0)  return atan(y/x);
    if (x < 0 && y >= 0)   return half_PI() + atan(-x/y);
    if (x < 0 && y < 0)  return PI() + atan(y/x);
    return half_PI() + PI() + atan(-x/y);
}

// Order angles 0->256 such that 192->0 and 64->255 to get a sensible priority order
int get_angle_priority(int angle) {
    // Rotate 90 degrees cw so 0 & 256 = furthest (most upwards)
    angle = (angle + 64)%256;

    // Normalise cw and ccw from 128 as distance from 128
    int distance_to = abs(128 - angle);

    distance_to *= 2;
    if (angle < 128) { distance_to -= 1; }

    return 256 - distance_to;
}

// Check in a circle around the cell to see if it has any other cells that will enter next frame.
// Return the lowest cell priority for entering the cell
int velocity_cell_priority(int x, int y, int width, int height, __global const short *src_pos, __global const short *src_vel) {
    // Radius is 16 as this is the maximum moved in 1 tick
    float search_radius = 16;
    float dist_squared = search_radius * search_radius;
    // Priority must be < 256
    int lowest_priority = 1000;
    // Iterate through each y of the circle
    for (int dx = -search_radius; dx <= search_radius; dx++) {
        for (int dy = - search_radius; dy <= search_radius; dy++) {
            if (dx*dx + dy*dy < dist_squared) {
                if (is_falling(x,y,width,height,src_pos)) {
                    int index = pos_to_index(dx + x,dy + y,width);
                    int vel_angle = src_vel[index / 3] & 255;
                    int vel_magnitude = src_vel[index / 3] & 65280 >> 8;

                    // Check if the cell will intersect x,y
                    // If it will, get its priority

                    float rad_angle = PI() * (float)vel_angle / 128.0f;
                    float frac_magnitude = vel_magnitude / 16.0f;

                    float x_vel = cos(rad_angle) * frac_magnitude;
                    float y_vel = sin(rad_angle) * frac_magnitude;

                    if ((int)(dx + x_vel) == x && (int)(dy + y_vel) == y) {
                        // A cell should only move to this cell if it has the lowest priority
                        int n_priority = get_angle_priority(vel_angle);
                        if (n_priority < lowest_priority) { lowest_priority = n_priority; }
                    }
                }
            }
        }
    }
    return lowest_priority;
}


// Update the velocity of the PosVel by the global acceleration factor
struct PosVel accelerate(struct PosVel pos_vel) {
    float rad_angle = PI() * (float)pos_vel.vel_angle / 128.0f;
    float frac_magnitude = pos_vel.vel_magnitude / 16.0f;
    float x_vel = cos(rad_angle) * frac_magnitude + get_x_accel();
    float y_vel = sin(rad_angle) * frac_magnitude + get_y_accel();

    rad_angle = get_angle(x_vel,y_vel);
    int byte_angle = (int)(128 * rad_angle / PI()) % 256;
    int byte_magnitude = (int)(16 * sqrt(x_vel*x_vel + y_vel * y_vel));

    if (byte_magnitude >= 256) { byte_magnitude = 255; }
    if (byte_angle >= 256) { byte_angle = 255; }

    pos_vel.vel_magnitude = byte_magnitude;
    pos_vel.vel_angle = byte_angle;

    return pos_vel;
}

// Return true if particle is near a static particle
bool near_static(struct PosVel pos_vel, int width, int height, __global const short *src_pos, __global const short *src_vel) {
    int tx, ty;

    // Search the 3x3 radius for a falling cell with 0 velocity
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            // Ignore own cell
            if (dx == 0 && dy == 0) { continue; }
            tx = dx + pos_vel.x;
            ty = dy + pos_vel.y;

            // Ignore invalid cells outside of world bounds
            if (tx < 0 || tx >= width || ty < 0 || ty >= height) { continue; }

            int index = pos_to_index(tx,ty,width);
            bool is_static = src_vel[index/3] == 0;
            if (!is_empty(tx,ty,width,height,src_pos) && is_static) {
                return true;
            }
        }
    }
    return false;
}

// Update the PosVel by checking which locations in the attempted line of movement can be accessed
// Hitting edge: set velocity to 0, both to 0 if its the bottom edge
// Hitting a static cell: if 0 velocity in a 3x3 around the cell, set velocity to 0
struct PosVel linear_check (struct PosVel pos_vel, int width, int height, __global const short *src_pos, __global const short *src_vel, __global short *dst_pos){
    // Get current position and target position by adding the vx and vy to the x and y
    // If target is invalid, confine it to the world bounds
    // Iterate across the line from source to target, checking for static or dynamic collisions

    // pos_vel = the start position (x1,y1)

    float rad_angle = PI() * (float)pos_vel.vel_angle / 128.0f;
    float frac_magnitude = pos_vel.vel_magnitude / 16.0f;
    float x_vel = cos(rad_angle) * frac_magnitude + get_x_accel();
    float y_vel = sin(rad_angle) * frac_magnitude + get_y_accel();

    // Get the target position (x2,y2)
    int target_x = pos_vel.x + x_vel;
    int target_y = pos_vel.y + y_vel;


    float dx = (target_x - pos_vel.x);
    float dy = (target_y - pos_vel.y);

    int line_step = abs((int)dx);
    if (abs((int)dy) > abs((int)dx)) { line_step = abs((int)dy); }

    // Amount to step each tick
    dx = dx / (float)line_step;
    dy = dy / (float)line_step;

    float x = (float)pos_vel.x;
    float y = (float)pos_vel.y;

    bool is_static = false;
    int i = 0;

    // Linear walk through each cell from source to target
    while (i <= line_step+1 && !is_static) {

        pos_vel.x = (int)x;
        pos_vel.y = (int)y;

        int index = pos_to_index(pos_vel.x,pos_vel.y,width);
        //dst_pos[index] = 30000;
        // Stop the line when static cell is found
        if (pos_vel.y <= 0 || pos_vel.y >= height - 1 || pos_vel.x <= 0 || pos_vel.x >= width -1 || near_static(pos_vel,width,height,src_pos,src_vel)) {
            pos_vel.vel_angle = 0;
            pos_vel.vel_magnitude = 0;
            is_static = true;
        }
        x += dx;
        y += dy;

        i++;
    }

    rad_angle = get_angle(x_vel,y_vel);

    return pos_vel;
}

// Move to the next free cell from the starting position in the given direction
struct PosVel update_velocity(struct PosVel pos_vel, int width, int height, __global const short *src_pos, __global const short *src_vel, __global short *dst_pos) {

    if (!near_static(pos_vel,width,height,src_pos,src_vel) && pos_vel.y > 0) {
        pos_vel = accelerate(pos_vel);
        pos_vel = linear_check(pos_vel,width,height,src_pos,src_vel,dst_pos);
    }

    return pos_vel;
}

__kernel void sampleKernel(__global const short *src_pos, __global const short *src_vel, __global short *dst_pos, __global short *dst_vel, __global const int *world_dims, __global int *step_ptr) {
    int gid = get_global_id(0);
    const int width = world_dims[0];
    const int height = world_dims[1];
    int x = (gid) % width;
    int y = (gid) / width;
    int step = step_ptr[0];


    // Clear the whole screen to remove junk data
    if (step == 0) {
        dst_pos[3 * gid] = 0;
        dst_pos[3 * gid + 1] = 0;
        dst_pos[3 * gid + 2] = 0;
        dst_vel[gid] = 0;
        return;
    }

    // Assign the RGB to cell_colour
    struct Colour cell_colour = get_cell(x,y,width,height,src_pos);

    // Default output value is blank
    // Mutate the cell_colour in the update step
    if (!is_empty(x,y,width,height,src_pos)) {
        short velocity = src_vel[gid];
		int vel_magnitude = ((velocity & 65280) >> 8);
		int vel_angle = (velocity & 255);

        struct PosVel pos_vel = {x,y,vel_magnitude,vel_angle};

        int new_index = pos_to_index(x,y,width);

        dst_vel[gid] = 0;

        // Move the cell
        dst_pos[3 * gid] = 0;
        dst_pos[3 * gid + 1] = 0;
        dst_pos[3 * gid + 2] = 0;

        //pos_vel = update_velocity(pos_vel,width,height,src_pos,src_vel,dst_pos);

        if (true || near_static(pos_vel,width,height,src_pos,src_vel)) {
            // Allow heavier particles to sink in lighter particles
            if (is_falling(pos_vel.x,pos_vel.y,width,height,src_pos)) { pos_vel = update_weight(pos_vel, width, height, step, src_pos); }

            // Fall and flow due to gravity
            if (is_powder(pos_vel.x,pos_vel.y,width,height,src_pos)) { pos_vel = update_powder(pos_vel,width,height,src_pos); }
            if (is_liquid(pos_vel.x,pos_vel.y,width,height,src_pos)) { pos_vel = update_liquid(pos_vel,width,height,step,src_pos); }

            if (pos_vel.x == -1) { return;}
        }

        velocity = pos_vel.vel_angle + (pos_vel.vel_magnitude << 8);

        new_index = pos_to_index(pos_vel.x,pos_vel.y,width);

        if (new_index == -1) { return; }


        dst_pos[new_index]     = cell_colour.r;
        dst_pos[new_index + 1] = cell_colour.g;
        dst_pos[new_index + 2] = cell_colour.b;
        dst_vel[new_index / 3] = velocity;

    }
};
