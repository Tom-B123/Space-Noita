#include "hip/hip_runtime.h"
// Holds RGB colour data for a pixel
struct Colour {
    // Stores the colour in a more readable format, chars used as channels must be between 0 and 31
    char r;
    char g;
    char b;
    bool a;
};

// Stores all the relevant data about a pixel and its behaviour
struct Tag {
    int     id;
    int     weight;
    bool    is_powder;
    bool    is_liquid;
    bool    is_gas;
    bool    is_flammable;
};

// Range of colours corresponds to the ID
struct Bound {
    int min_r;
    int max_r;
    int min_g;
    int max_g;
    int min_b;
    int max_b;
    int id;
};

struct Colour get_pixel_colour(short pixel_data) {
    struct Colour out = {(char)(pixel_data & 63488) >> 11, (char)(pixel_data & 1984) >> 6, (char)(pixel_data & 62) >> 1, (bool)(pixel_data & 1) };
    return out;
}
short get_pixel_data(struct Colour pixel_colour) {
    short out = (short)(pixel_colour.a + (pixel_colour.b << 1) + (pixel_colour.g << 6) + (pixel_colour.r << 11));
    return out;
}

int get_material_id(struct Colour pixel_colour, struct Bound *bounds) {
    for (int i = 0; i < 2; i++) {
        if (pixel_colour.r > 0) printf("min: %i, max: %i, pixel: %i \n",bounds[i].min_r,bounds[i].max_r,pixel_colour.r);
        if (
            pixel_colour.r >= bounds[i].min_r && pixel_colour.r <= bounds[i].max_r &&
            pixel_colour.g >= bounds[i].min_g && pixel_colour.g <= bounds[i].max_g &&
            pixel_colour.b >= bounds[i].min_b && pixel_colour.b <= bounds[i].max_b
        ) {
            return i;
        }
    }
    return -1;
}

struct Tag get_tags(struct Colour pixel_colour, struct Bound *bounds) {
    int material_id = get_material_id(pixel_colour,bounds);

    if (material_id > -1) printf("%i \n",material_id);

    struct Tag out_tag = {material_id,0,false,false,false,false};
    switch(material_id) {
        default:
            return out_tag;
    }
}

__kernel void sampleKernel(__global const short *src_pos, __global short *dst_pos,__global const int *world_dims, __global int *step_ptr) {
    const int gid = get_global_id(0);
    const int width = world_dims[0];
    const int height = world_dims[1];
    const int x = (gid) % width;
    const int y = (gid) / width;
    const int step = step_ptr[0];

    struct Colour cell_colour = get_pixel_colour(src_pos[gid]);

    struct Bound bounds[] = {
        {29,31,29,31,0,0,0}, // Sand (ID 0)
        {0,0,0,2,29,31,1},   // Water (ID 1)
    };

    if (cell_colour.r == 0)printf("%i,%i,%i \n",cell_colour.r,cell_colour.g,cell_colour.b);

    struct Tag tag = get_tags(cell_colour,&bounds);

    dst_pos[gid] = get_pixel_data(cell_colour);
    return;
};
