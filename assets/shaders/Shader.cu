#include "hip/hip_runtime.h"
// Holds RGB colour data for a pixel
struct Colour {
    // Stores the colour in a more readable format, chars used as channels must be between 0 and 31
    short r;
    short g;
    short b;
    bool a;
};

// Stores all the relevant data about a pixel and its behaviour
struct Tag {
    int     id;
    int     weight;
    bool    is_powder;
    bool    is_liquid;
    bool    is_gas;
    bool    is_flammable;
};

// Range of colours corresponds to the ID
struct Bound {
    int min_r;
    int max_r;
    int min_g;
    int max_g;
    int min_b;
    int max_b;
    int id;
};

struct Pos {
    int x;
    int y;
};

float PI() { return 3.141592654f; }
float half_PI() { return 1.570796327f; }

int pos_to_index(struct Pos pos, int width) {
    return (pos.x+pos.y*width);
}

int get_gravity_direction(float src_gravity_angle,int step) {
    // Split 8 directions into n more "pseudo directions"
    // High n = obvious alternation, low n = low degree of extra directions
    const int increments = 4;

    // Step of the simulation within these increments
    int sub_step = step % increments;

    float direction = 8 * src_gravity_angle / 2 / PI();

    // 2 direction options
    int remain_direction = (int)(floor(direction)) % 8;
    int change_direction = (remain_direction + 1) % 8;

    // Fractional component between the remain and change directions
    float decimal = direction - floor(direction);

    int remain_weight = (int)(floor((1-decimal) * increments + 0.5f));

    if (sub_step <= remain_weight) { return remain_direction; }
    return change_direction;
}

struct Colour get_pixel_colour(short pixel_data) {
    int i_pixel_data = (int)pixel_data;
    struct Colour out = {(short)(i_pixel_data & 63488) >> 11, (short)(i_pixel_data & 1984) >> 6, (short)(i_pixel_data & 62) >> 1, (bool)(pixel_data & 1) };

    if (out.r < 0) { out.r += 32; }

    return out;
}
short get_pixel_data(struct Colour pixel_colour) {
    short out = (short)(pixel_colour.a + (pixel_colour.b << 1) + (pixel_colour.g << 6) + (pixel_colour.r << 11));
    return out;
}

int get_material_id(struct Colour pixel_colour, struct Bound *bounds) {
    for (int i = 0; i < 2; i++) {
        //if (pixel_colour.r > 0) printf("min: %i, max: %i, pixel: %i \n",bounds[i].min_r,bounds[i].max_r,pixel_colour.r);
        if (
            pixel_colour.r >= bounds[i].min_r && pixel_colour.r <= bounds[i].max_r &&
            pixel_colour.g >= bounds[i].min_g && pixel_colour.g <= bounds[i].max_g &&
            pixel_colour.b >= bounds[i].min_b && pixel_colour.b <= bounds[i].max_b
        ) {
            return i;
        }
    }
    return -1;
}

struct Tag get_tags(struct Colour pixel_colour, struct Bound *bounds) {
    int material_id = get_material_id(pixel_colour,bounds);

    struct Tag out_tag = {material_id,0,false,false,false,false};
    switch(material_id) {
        case 0:
            // Sand
            out_tag.is_powder = true;
            break;
        case 1:
            // Water
            out_tag.is_liquid = true;
            break;
        default:
            break;
    }
    return out_tag;
}

bool is_falling(struct Pos pos, int width, struct Bound *bounds, __global const short *src_pos) {
    struct Tag tag = get_tags(get_pixel_colour(src_pos[pos_to_index(pos,width)]),bounds);
    return tag.is_powder || tag.is_liquid;
}

bool is_empty(struct Pos pos, int width, int height, __global const short *src_pos) {
    struct Colour cell_colour = get_pixel_colour(src_pos[pos_to_index(pos,width)]);
    if (pos.x < 0 || pos.x > width - 1 || pos.y < 0 || pos.y > height - 1) { return false; }
    return !cell_colour.a;
}

int falling_cell_priority(struct Pos pos, int width, float gravity_angle, int step, struct Bound *bounds, __global const short *src_pos) {

    int gravity_direction = get_gravity_direction(gravity_angle,step);

    int offset_x[] = {
         0,  1,  1,  1,  0, -1, -1, -1,
    };

    int offset_y[] = {
        -1, -1,  0,  1,  1,  1,  0, -1,
    };

    // Cell falling from above
    for (int i = 0; i < 5; i++) {
        // Look at cell moving into the position
        pos.x -= offset_x[gravity_direction];
        pos.y -= offset_y[gravity_direction];

        if (is_falling(pos,width,bounds,src_pos)) { return i;}

        pos.x += offset_x[gravity_direction];
        pos.y += offset_y[gravity_direction];

        // Move gravity_direction to the next cell
        gravity_direction = gravity_direction + ((i+1) * (2 * ((i+1)%2) - 1))%8;
        if (gravity_direction < 0) {gravity_direction += 8;}
    }

    return 4;
}

struct Pos update_powder(struct Pos pos, int width, int height, float gravity_angle, int step, struct Bound *bounds, __global const short *src_pos) {

    int initial_x = pos.x;
    int initial_y = pos.y;

    int gravity_direction = get_gravity_direction(gravity_angle,step);

    int offset_x[] = {
         0,  1,  1,  1,  0, -1, -1, -1,
    };

    int offset_y[] = {
        -1, -1,  0,  1,  1,  1,  0, -1,
    };

    for (int i = 0; i < 3; i++) {
        // Look at cell moving into the position
        pos.x += offset_x[gravity_direction];
        pos.y += offset_y[gravity_direction];

        if (is_empty(pos,width,height,src_pos) && falling_cell_priority(pos,width,gravity_angle,step,bounds,src_pos) >= i) { return pos;}

        pos.x -= offset_x[gravity_direction];
        pos.y -= offset_y[gravity_direction];

        // Move gravity_direction to the next cell
        gravity_direction = gravity_direction + ((i+1) * (2 * ((i+1)%2) - 1));
        if (gravity_direction < 0) {gravity_direction += 8;}
    }

    pos.x = initial_x;
    pos.y = initial_y;

    return pos;
}

struct Pos update_liquid(struct Pos pos, int width, int height, float gravity_angle, int step, struct Bound *bounds, __global const short *src_pos) {

    int initial_x = pos.x;
    int initial_y = pos.y;

    int gravity_direction = get_gravity_direction(gravity_angle,step);

    int offset_x[] = {
         0,  1,  1,  1,  0, -1, -1, -1,
    };

    int offset_y[] = {
        -1, -1,  0,  1,  1,  1,  0, -1,
    };

    for (int i = 0; i < 5; i++) {
        // Look at cell moving into the position
        pos.x += offset_x[gravity_direction];
        pos.y += offset_y[gravity_direction];

        if (is_empty(pos,width,height,src_pos) && falling_cell_priority(pos,width,gravity_angle,step,bounds,src_pos) >= i) { return pos;}

        pos.x -= offset_x[gravity_direction];
        pos.y -= offset_y[gravity_direction];

        // Move gravity_direction to the next cell
        gravity_direction = gravity_direction + ((i+1) * (2 * ((i+1)%2) - 1));
        if (gravity_direction < 0) {gravity_direction += 8;}
    }

    pos.x = initial_x;
    pos.y = initial_y;

    return pos;
}

__kernel void sampleKernel(__global const short *src_pos, __global short *dst_pos, __global const float *src_gravity_angle, __global const int *src_world_dims, __global int *src_step) {
    const int gid = get_global_id(0);
    const int width = src_world_dims[0];
    const int height = src_world_dims[1];
    const int x = (gid) % width;
    const int y = (gid) / width;
    const int step = src_step[0];
    const float gravity_angle = src_gravity_angle[0];

    const struct Colour blank = { 0,0,0,0 };

    struct Pos pos = {x,y};

    struct Colour cell_colour = get_pixel_colour(src_pos[gid]);

    // Materials should have a range, eg. 2 in rgb. These colours correspond to unique colours, e.g. metal can be white / red etc.
    struct Bound bounds[] = {
        {29,31,29,31, 0, 2, 0}, // Sand (ID 0)
        { 0, 2, 0, 2,29,31, 1},   // Water (ID 1)
    };

    struct Tag tag = get_tags(cell_colour,&bounds);

    if (tag.is_powder) { pos = update_powder(pos,width,height,gravity_angle,step,&bounds,src_pos); }
    if (tag.is_liquid) { pos = update_liquid(pos,width,height,gravity_angle,step,&bounds,src_pos); }

    int n_id = pos_to_index(pos,width);

    dst_pos[gid] = get_pixel_data(blank);
    dst_pos[n_id] = get_pixel_data(cell_colour);
    return;
};
